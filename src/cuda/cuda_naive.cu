#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include "../../lib/hip/hip_runtime_api.h"

#define REPETITIONS 50

__global__
void device_transpose(int size, int* mat){
    for(int i = 0; i < size; i++){
        for(int j = i+1; j < size; j++){
            int tmp = mat[i*size+j];
            mat[i*size+j] = mat[j*size+i];
            mat[j*size+i] = tmp;
        }
    }
}

void host_transpose(int size, int* mat){
    for(int i = 0; i < size; i++){
        for(int j = i+1; j < size; j++){
            int tmp = mat[i*size+j];
            mat[i*size+j] = mat[j*size+i];
            mat[j*size+i] = tmp;
        }
    }
}

/*
    Compute the effective bandwidth of a matrix transposition. 

    params:
        size: Size of the matrix
        time: Time needed to transpose the matrix
    return:
        The effective bandwidth in bytes
*/
double compute_effective_bandwidth(int size, double time){
    int num_matrix_elements = size * size;
    int matrix_size_in_bytes = num_matrix_elements * sizeof(int);
    // Size we read and write every element, the amount of moved bytes is twice the total bytes of the matrix
    double moved_bytes = 2.0*matrix_size_in_bytes; 
    return moved_bytes / time;
}

/*
    Print the matrix on the terminal.

    params:
        size: Size of the matrix
        mat: Matrix of initialize
    return:
        void
*/
void print_matrix(int size, int* mat){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            printf("%2d ", mat[i*size + j]);
        }
        printf("\n");
    }
}

/*
    Print information for debugging on the terminal.

    params:
        size: Size of the matrix
        mat: Transposed matrix
        time: Time needed for transposition
    return:
        void
*/
void print_debug_info(int size, int *mat, double time, double bandwidth){
    for(int i = 0; i < 3*size; i++){
        printf("-");    
    }
    printf("\n");
    print_matrix(size, mat);
    printf("\n");
    printf("Time for transposition: %f\n", time);
    printf("Effective bandwidth: %.15f\n", bandwidth);
}

/*
    Initialize matrix with random values between 0 and 99.

    params:
        size: Size of the matrix
        mat: Transposed matrix
    return:
        void
*/
void init_matrix(int size, int* mat){
    for(int i = 0; i < size * size; i++){
        mat[i] = rand() % 100;
    }
}

int main(int argc, char **argv){
    
    // Check if argumnet is present
    if(argc < 2){
        printf("One argument expected. But got %d arguments.\n", argc-1);
        exit(EXIT_FAILURE);
    }

    // Check if debug mode is activated
    bool debug_mode = false;
    if(argc >= 3 && strcmp(argv[2], "--debug") == 0){
        debug_mode = true;
    }
    
    // Compute matrix size
    int input = atoi(argv[1]);
    if(input == 0){
        printf("Provide a valid numeric value. Not '%s'\n", argv[1]);
        exit(EXIT_FAILURE);
    }
    int size = (int) pow(2.0, input);
    
    // Allocate matrix memory on host and device
    int* host_mat = (int *) malloc(size * size * sizeof(int));
    int* dev_mat;
    checkCudaErrors( hipMalloc(&dev_mat, size * size * sizeof(int)) );

    // Init matrix
    init_matrix(size, host_mat);

    // Time host
    for(int t=0; t < REPETITIONS; t++){
        init_matrix(size, host_mat);
        if(debug_mode){
            print_matrix(size, host_mat);
        }
        clock_t begin = clock();
        host_transpose(size, host_mat);
        clock_t end = clock();
        double time = (double)(end - begin) / CLOCKS_PER_SEC;
        double bandwidth = compute_effective_bandwidth(size, time);
        if(debug_mode){
            print_debug_info(size, host_mat, time, bandwidth);
        }
        else {
            printf("%f,%f,host,0,0\n", time, bandwidth);    
        }
    }

    // Time device
    for(int t=0; t < REPETITIONS; t++){
        init_matrix(size, host_mat);
        if(debug_mode){
            print_matrix(size, host_mat);
        }

        clock_t begin = clock();
        hipMemcpy(dev_mat, host_mat, size * size * sizeof(int), hipMemcpyHostToDevice);
        device_transpose<<<1,1>>>(size, dev_mat);
        checkCudaErrors( hipDeviceSynchronize() );
        hipMemcpy(host_mat, dev_mat, size * size * sizeof(int), hipMemcpyDeviceToHost);
        clock_t end = clock();

        double time = (double)(end - begin) / CLOCKS_PER_SEC;
        double bandwidth = compute_effective_bandwidth(size, time);
        if(debug_mode){
            print_debug_info(size, host_mat, time, bandwidth);
        }
        else {
            printf("%f,%f,device,0,0\n", time, bandwidth);    
        }
    }

    hipFree(dev_mat);
    free(host_mat);

    return 0;
}