#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include "../../lib/hip/hip_runtime_api.h"


#define REPETITIONS 50
#define TILE_DIM 32

__global__ void transpose(int size, int* matrix) {
    __shared__ int tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int i = 0; i < TILE_DIM; i += blockDim.y) {
        if (x < size && y + i < size) {
            tile[threadIdx.y + i][threadIdx.x] = matrix[(y + i) * size + x];
        }
    }

    __syncthreads();

    for (int i = 0; i < TILE_DIM; i += blockDim.y) {
        if (x < size && y + i < size) {
            matrix[(y + i) * size + x] = tile[threadIdx.x][threadIdx.y + i];
        }
    }
}

/*
    Compute the effective bandwidth of a matrix transposition. 

    params:
        size: Size of the matrix
        time: Time needed to transpose the matrix
    return:
        The effective bandwidth in bytes
*/
float compute_effective_bandwidth(int size, float time){
    int num_matrix_elements = size * size;
    int matrix_size_in_bytes = num_matrix_elements * sizeof(int);
    // Size we read and write every element, the amount of moved bytes is twice the total bytes of the matrix
    float moved_bytes = 2.0*matrix_size_in_bytes; 
    return moved_bytes / time;
}

/*
    Print the matrix on the terminal.

    params:
        size: Size of the matrix
        mat: Matrix of initialize
    return:
        void
*/
void print_matrix(int size, int* mat){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            printf("%2d ", mat[i*size + j]);
        }
        printf("\n");
    }
}

/*
    Print information for debugging on the terminal.

    params:
        size: Size of the matrix
        mat: Transposed matrix
        time: Time needed for transposition
    return:
        void
*/
void print_debug_info(int size, int *mat, double time, double bandwidth){
    for(int i = 0; i < 3*size; i++){
        printf("-");    
    }
    printf("\n");
    print_matrix(size, mat);
    printf("\n");
    printf("Time for transposition: %f\n", time);
    printf("Effective bandwidth: %.15f\n", bandwidth);
}

/*
    Initialize matrix with random values between 0 and 99.

    params:
        size: Size of the matrix
        mat: Transposed matrix
    return:
        void
*/
void init_matrix(int size, int* mat){
    for(int i = 0; i < size * size; i++){
        mat[i] = rand() % 100;
    }
}

int main(int argc, char** argv) {
    // Check if argumnet is present
    if(argc < 2){
        printf("One argument expected. But got %d arguments.\n", argc-1);
        exit(EXIT_FAILURE);
    }

    // Check if debug mode is activated
    bool debug_mode = false;
    if(argc >= 3 && strcmp(argv[2], "--debug") == 0){
        debug_mode = true;
    }
    
    // Compute matrix size
    int input = atoi(argv[1]);
    if(input == 0){
        printf("Provide a valid numeric value. Not '%s'\n", argv[1]);
        exit(EXIT_FAILURE);
    }
    int size = (int) pow(2.0, input);
    
    // Allocate matrix memory on host and device
    int* host_mat = (int *) malloc(size * size * sizeof(int));
    int* dev_mat;
    checkCudaErrors( hipMalloc(&dev_mat, size * size * sizeof(int)) );
    
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid((size + TILE_DIM - 1) / TILE_DIM, (size + TILE_DIM - 1) / TILE_DIM);
    // Init matrix
    init_matrix(size, host_mat);

    // warm up
    checkCudaErrors( hipMemcpy(dev_mat, host_mat, size * size * sizeof(int), hipMemcpyHostToDevice) );
    transpose<<<dimGrid, dimBlock>>>(size, dev_mat);
    checkCudaErrors( hipDeviceSynchronize() );
    checkCudaErrors( hipMemcpy(host_mat, dev_mat, size * size * sizeof(int), hipMemcpyDeviceToHost) );

    // Time device
    for(int t=0; t < REPETITIONS; t++){
        init_matrix(size, host_mat);
        if(debug_mode){
            print_matrix(size, host_mat);
        }

        checkCudaErrors( hipMemcpy(dev_mat, host_mat, size * size * sizeof(int), hipMemcpyHostToDevice) );
        clock_t begin = clock();
        transpose<<<dimGrid,dimBlock>>>(size, dev_mat);
        checkCudaErrors( hipDeviceSynchronize() );
        clock_t end = clock();
        checkCudaErrors( hipMemcpy(host_mat, dev_mat, size * size * sizeof(int), hipMemcpyDeviceToHost) );
        float time = (float)(end - begin) / CLOCKS_PER_SEC;
    
        float bandwidth = compute_effective_bandwidth(size, time);
        if(debug_mode){
            print_debug_info(size, host_mat, time, bandwidth);
        }
        else {
            printf("%f,%f,gpu_tiled\n", time, bandwidth);    
        }
    }

    checkCudaErrors( hipFree(dev_mat) );
    free(host_mat);

    return 0;
}
