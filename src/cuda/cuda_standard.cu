#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include "../../lib/hip/hip_runtime_api.h"

#define REPETITIONS 50

__global__ void transpose(int *matrix, int size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < size && y < size && x < y) {
        int idx1 = y * size + x;
        int idx2 = x * size + y;
        int temp = matrix[idx1];
        matrix[idx1] = matrix[idx2];
        matrix[idx2] = temp;
    }
}

/*
    Initialize matrix with random values between 0 and 99.

    params:
        size: Size of the matrix
        mat: Transposed matrix
    return:
        void
*/
void init_matrix(int size, int* mat){
    for(int i = 0; i < size * size; i++){
        mat[i] = rand() % 100;
    }
}

/*
    Print the matrix on the terminal.

    params:
        size: Size of the matrix
        mat: Matrix of initialize
    return:
        void
*/
void print_matrix(int size, int* mat){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            printf("%2d ", mat[i*size + j]);
        }
        printf("\n");
    }
}

/*
    Print information for debugging on the terminal.

    params:
        size: Size of the matrix
        mat: Transposed matrix
        time: Time needed for transposition
    return:
        void
*/
void print_debug_info(int size, int *mat, double time, double bandwidth){
    for(int i = 0; i < 3*size; i++){
        printf("-");    
    }
    printf("\n");
    print_matrix(size, mat);
    printf("\n");
    printf("Time for transposition: %f\n", time);
    printf("Effective bandwidth: %.15f\n", bandwidth);
}

/*
    Compute the effective bandwidth of a matrix transposition. 

    params:
        size: Size of the matrix
        time: Time needed to transpose the matrix
    return:
        The effective bandwidth in bytes
*/
double compute_effective_bandwidth(int size, double time){
    int num_matrix_elements = size * size;
    int matrix_size_in_bytes = num_matrix_elements * sizeof(int);
    // Size we read and write every element, the amount of moved bytes is twice the total bytes of the matrix
    double moved_bytes = 2.0*matrix_size_in_bytes; 
    return moved_bytes / time;
}

int main(int argc, char **argv) {

    // Check if argumnet is present
    if(argc < 2){
        printf("One argument expected. But got %d arguments.\n", argc-1);
        exit(EXIT_FAILURE);
    }

    // Check if debug mode is activated
    bool debug_mode = false;
    if(argc >= 3 && strcmp(argv[2], "--debug") == 0){
        debug_mode = true;
    }
    
    // Compute matrix size
    int input = atoi(argv[1]);
    if(input == 0){
        printf("Provide a valid numeric value. Not '%s'\n", argv[1]);
        exit(EXIT_FAILURE);
    }
    int size = (int) pow(2.0, input);

    int* h_matrix = (int *) malloc(size * size * sizeof(int));
    init_matrix(size, h_matrix);

    int *d_matrix;
    hipMalloc(&d_matrix, size * size * sizeof(int));

    dim3 blockSize(16, 16);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y);

    for(int t=0; t < REPETITIONS; t++){
        init_matrix(size, h_matrix);
        if(debug_mode){
            print_matrix(size, h_matrix);
        }
        clock_t begin = clock();
        hipMemcpy(d_matrix, h_matrix, size * size * sizeof(int), hipMemcpyHostToDevice);
        transpose<<<gridSize, blockSize>>>(d_matrix, size);
        hipDeviceSynchronize();
        hipMemcpy(h_matrix, d_matrix, size * size * sizeof(int), hipMemcpyDeviceToHost);
        clock_t end = clock();
        double time = (double)(end - begin) / CLOCKS_PER_SEC;
        double bandwidth = compute_effective_bandwidth(size, time);
        if(debug_mode){
            print_debug_info(size, h_matrix, time, bandwidth);
        }
        else {
            printf("%f,%f,device,0,0\n", time, bandwidth);    
        }
    }

    hipFree(d_matrix);
    free(h_matrix);

    return 0;
}
